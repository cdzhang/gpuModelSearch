#include <stdio.h>
#include <stdlib.h>
#include <R.h>

extern "C" void gpuReset( int * out );

void gpuReset(int * out){
  hipError_t err;

  err = hipDeviceReset();

  if(hipSuccess == err)
    *out = 0;
  else
    *out = 1;
}



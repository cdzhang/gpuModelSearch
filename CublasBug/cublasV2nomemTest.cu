
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

int main(void){
  int i;
  for(i = 0; i < 100000; i++){
    hipblasHandle_t han;
    
    printf(" %d \n", i);

    hipblasCreate(&han);
    
    hipblasDestroy(han);

  }
  return 0;
}

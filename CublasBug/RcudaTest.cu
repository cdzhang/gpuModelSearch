#include <stdio.h>
#include <stdlib.h>
#include <R.h>

extern "C" void Rcuda(int *n){
  float * dX;

  hipMalloc((void **)&dX, (*n)*sizeof(float));

  hipFree(dX);


}

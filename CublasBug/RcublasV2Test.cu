#include <stdio.h>
#include <stdlib.h>
#include <R.h>
#include <hipblas.h>

extern "C" void RcublasV2(int *n){
  float * dX;
  hipblasHandle_t han;

  hipblasCreate(&han);

  hipMalloc((void **)&dX, (*n)*sizeof(float));

  hipFree(dX);

  hipblasDestroy(han);


}

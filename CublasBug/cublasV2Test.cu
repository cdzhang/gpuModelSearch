
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>

int main(void){
  int i;
  for(i = 0; i < 100000; i++){
    float * dX;
    hipblasHandle_t han;
    
    printf(" %d \n", i);

    hipblasCreate(&han);
    
    hipMalloc((void **)&dX, 10*sizeof(float));
    
    hipFree(dX);
    
    hipblasDestroy(han);

  }
  return 0;
}
